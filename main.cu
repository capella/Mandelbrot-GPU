#include "hip/hip_runtime.h"
#include <GLFW/glfw3.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define WIDTH 700*2
#define HEIGHT 500*2

GLuint buffer, pboTextureId;
struct hipGraphicsResource* buffer_CUDA;

int g_width = WIDTH;
int g_height = HEIGHT;

const int IterationMax=200;
const double EscapeRadius=2;
const double ER2=EscapeRadius*EscapeRadius;
double base_x = -2.5;
double base_y = -2.0;
double zoom = 0.003;

void deleteVBO() { 
    hipGraphicsUnregisterResource(buffer_CUDA);
    glDeleteBuffers(1, &buffer);
}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    double tmp, tmp2;
    if (key == GLFW_KEY_D) {
        base_x+=40*zoom;
    } else if (key == GLFW_KEY_A) {
        base_x-=40*zoom;
    } else if (key == GLFW_KEY_W) {
        base_y+=40*zoom;
    } else if (key == GLFW_KEY_S) {
        base_y-=40*zoom;
    } else if (key == GLFW_KEY_I) {
        tmp = g_width * zoom;
        tmp2 = g_height * zoom;
        zoom*=1.6;
        tmp -= g_width * zoom;
        tmp2 -= g_height * zoom;
        base_x += tmp/2.0;
        base_y += tmp2/2.0;
    } else if (key == GLFW_KEY_O) {
        tmp = g_width * zoom;
        tmp2 = g_height * zoom;
        zoom/=1.6;
        tmp -= g_width * zoom;
        tmp2 -= g_height * zoom;
        base_x += tmp/2.0;
        base_y += tmp2/2.0;
    }
}

__global__ void createVertices(float4* positions, double base_x_in, double base_y_in, double zoom_in, unsigned int width) { 


    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate uv coordinates 

    // Write positions 
    int i;
    double Cy = base_y_in + y*zoom_in;
    double Cx = base_x_in + x*zoom_in;

    double Zx = 0.0;
    double Zy = 0.0;
    double Zx2 = Zx*Zx;
    double Zy2 = Zy*Zy;
    /* */
    for (i=0;i<IterationMax && ((Zx2+Zy2)<ER2);i++) {
        Zy=2*Zx*Zy + Cy;
        Zx=Zx2-Zy2 +Cx;
        Zx2=Zx*Zx;
        Zy2=Zy*Zy;
    }
    /* compute  pixel color (24 bit = 3 bytes) */
    if (i==IterationMax) { /*  interior of Mandelbrot set = black */
        positions[y * width + x] = make_float4(0, 0, 0, 1.0f);
    } else { /* exterior of Mandelbrot set = white */
        if (i < 10)
            positions[y * width + x] = make_float4(1, 1, i/10-1, 1.0f);
        else if (i < 30)
            positions[y * width + x] = make_float4((i-10)/20-1, (i-10)/20-1, 1, 1.0f);
        else if (i < 70)
            positions[y * width + x] = make_float4(1, (i-30)/40-1, 1, 1.0f);
        else if (i < 110)
            positions[y * width + x] = make_float4((i-70)/30-1, (i-70)/30-1, 1, 1.0f);
        else if (i < 170)
            positions[y * width + x] = make_float4((i-110)/80-1, 1, 1, 1.0f);
        else
            positions[y * width + x] = make_float4((i-170)/30-1, 1, (i-170)/30-1, 1.0f);
    }



    /*positions[y * width + x] = make_float4(1, 0, 0, 1.0f);*/
}


void create_buffer_and_texture () {
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, buffer);
    unsigned int size = g_width * g_height * 4 * sizeof(float);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, size, NULL, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    hipGraphicsGLRegisterBuffer(&buffer_CUDA, buffer, cudaGraphicsMapFlagsWriteDiscard);
}

void window_size_callback(GLFWwindow* window, int width, int height) {
    g_height = height;
    g_width = width;
    deleteVBO();
    create_buffer_and_texture ();
}

int main(void) {
    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(WIDTH/2, HEIGHT/2, "Hello World", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }

    glfwSetWindowSizeCallback(window, window_size_callback);
    glfwSetKeyCallback(window, key_callback);


    /* Make the window's context current */
    glfwMakeContextCurrent(window);

    hipSetDevice(0);

    glGenTextures(1, &pboTextureId);
    glBindTexture(GL_TEXTURE_2D, pboTextureId); // <== Bind the texture object!!
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, WIDTH, HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

    glGenBuffers(1, &buffer);
    create_buffer_and_texture ();

    glActiveTexture(GL_TEXTURE0);
    // This code is using the immediate mode texture object 0. Add an own texture object if needed.
    glBindTexture(GL_TEXTURE_2D, 0); // Just use the immediate mode texture.
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

    glTexEnvi(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE); // Not a texture. default is modulate.


    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window)) {

        float4* positions;
        hipGraphicsMapResources(1, &buffer_CUDA, 0);
        size_t num_bytes;
        hipGraphicsResourceGetMappedPointer((void**)&positions, &num_bytes, buffer_CUDA);
        // Execute kernel 
        dim3 dimBlock(16, 16, 1);
        dim3 dimGrid(g_width / dimBlock.x, g_height / dimBlock.y, 1);

        createVertices<<<dimGrid, dimBlock>>>(positions, base_x, base_y, zoom, g_width);
        // Unmap buffer object 
        hipGraphicsUnmapResources(1, &buffer_CUDA, 0);
        // Render from buffer object 

        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, buffer);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, g_width, g_height, 0, GL_RGBA, GL_FLOAT, NULL);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

        glEnable(GL_TEXTURE_2D);

        glBegin(GL_QUADS);
            glTexCoord2f(0.0f, 0.0f);
            glVertex2f(-1.0f, -1.0f);
            glTexCoord2f(1.0f, 0.0f);
            glVertex2f(1.0f, -1.0f);
            glTexCoord2f(1.0f, 1.0f);
            glVertex2f(1.0f, 1.0f);
            glTexCoord2f(0.0f, 1.0f);
            glVertex2f(-1.0f, 1.0f);
        glEnd();

        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }

    glfwTerminate();
    return 0;
}